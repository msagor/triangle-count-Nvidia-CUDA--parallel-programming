#include "hip/hip_runtime.h"
/*--------------------------------------------------------------------------- */
/* tri_gpu: compute the number of triangles in a graph (GPU method) */
/*--------------------------------------------------------------------------- */

// READ THIS:
// This code is way over-commented because I'm giving you lots of instruction
// on how to write a CUDA kernel and its CPU driver.  Please delete ALL C++
// style comments in this file (and only this file) that used the '//' comment
// style!  Replace them with your own that describe how you solved each part of
// this problem.  Keep the comments in the old style /* like this */, since
// those are useful.  Feel free to rewrite those /*comments*/ if you like.

#include "tri_def.h"

// I recommend using a 2D array of threads, x-by-y, since you have two nested
// for loops in the code below.  I recommend a single 1D array of threadblocks.
// Each threadblock must do only one column (or node) j at a time, since it
// needs to use the Mark array of size n to mark the neighbors of j, for use in
// the two nested loops.  I will let you figure out the dimensions to use.  If
// you are having trouble getting the code to work, try one block with a single
// thread (1-by-1 thread grid).  You won't have any synchronization problems,
// but of course your code will be exceedingly slow.

// However, if you want to use a 1D array of threads, feel free to do so.  Just
// be sure to keep things parallel.  Don't force one thread in the threadblock
// to do just one iteration of the "for (p = ...)" iteration below, for
// example.  That will painfully be slow (and points taken off for a bad
// algorithm).

// NBLOCKS needs to be large enough to keep the 13 SMs on GPU busy.  Don't make
// NBLOCKS too high, however.  Your kernel will need a Marks array of size
// NBLOCKS*n, so that each threadblock and have its own private Mark arry of
// size n.  If NBLOCKS is high you will use all the GPU memory for the Marks
// array, and you won't be able to solve the problems on the GPU.

/* -------------------------------------------------------------------------- */
/* tri_kernel: GPU kernel */
/* -------------------------------------------------------------------------- */

/* launched with <<<NBLOCKS, dim3(NX,NY)>>> */
// or modify it to launch with <<<NBLOCKS,NTHREADS>>> as you prefer
#define NBLOCKS 512 //TODO           /* gridDim.x                                */
#define NX 28	    //TODO           /* blockDim.x (# of threads in x dimension) */
#define NY 29	    //TODO           /* blockDim.y (# of threads in y dimension) */
#define NTHREADS (NY * NX)

__global__ void tri_kernel
(
    /* inputs, not modified: */
    const int *Ap,              /* column pointers, size n+1        */
    const int *Ai,              /* row indices                      */
    const int n,                /* A has n ndoes                    */
    /* workspace */
    bool *Marks,                /* size NBLOCKS*n so each threadblock has */
                                /* its own array of size n                */
    /* output: */
    int64_t Ntri_result [NBLOCKS] /* # triangles found by each threadblock */
)
{

    //POINTING THE Mark VARIABLE TO THE RIGHT POSITIONING OF THE MARKS ARRAY FOR THAT PARTICULAR BLOCKID
    bool *Mark = Marks + (n*blockIdx.x);	//TODO

    
	//CREATING A GLOBAL ID FOR EACH THREAD IN A BLOCK. WE ARE GOING TO ASSIGIN
	//THE FOR LOOP GOES FROM 0 TO n AND INCREMENTS BY NTHREADS.
	//ITS NOT THAT ALL THE THREADS IN A BLOCK WORK IN SEQUENCE, RATHER EACH THREAD WILL GRAB HIS OWN ID AND DO ITS JOB
	//HERE, WE ARE EMPTYING OUT THE MARK ARRAY FOR THAT SPECIFIC BLOCKID
    int id =  threadIdx.y * blockDim.x + threadIdx.x ; 	 //TODO
    for (int i = id ; i < n ; i+=NTHREADS)   //TODO
    {
        Mark [i] = 0 ;
    }


    /* ensure all threads have cleared the Mark array */
    // What happens if some threads in this threadblock finish clearing their
    // part of Mark, and then they start the work below before some other
    // threads have finished clearing their part of Mark?  Race condition!  I
    // put this sync threads here for you.  You will need more elsewhere in
    // this kernel.  I will let you figure out where.  When in doubt, extra
    // syncthreads are not usually harmful to code correctness (too many can
    // slow your code down however).
    __syncthreads ( ) ;

    /* each thread counts its own triangles in nt */
    // This variable is local to each thread.
    int64_t nt = 0 ;

    /* count the triangles for node j, aka A(:,j) of the adjacency matrix */
    //ASSIGINING EACH BLOCKS JOB FOR EACH COLUMN OF MATRIX TO DO THE JOB 
	//IF THE BLOCK NUMBER IS SMALLER THAN COLUMN NUMBER THEN SOME BLOCK(FULL OF THREADS) JUST NEVER WORK
	//ALSO EACH BLOCK DOES WORK ON EVERY NBLOCKS-th BLOCK AND INCREMENTS BY THE NUMBER OF BLOCKS
    for (int j = blockIdx.x  ; j < n; j+=NBLOCKS)  //TODO
    {

        /* All threads in a threadblock are working on this node j, */
        /* equivalently, column A(:,j) of the adjacency matrix A */

       
        /* scatter A(:,j) into Mark, marking all nodes adjacent to node j */
		//THIS FOR LOOP, JUST PICKS UP THE INDEX REGION IT NEEDS TO ITERATE OVER
		//FOR EACH J, WE GO TO THE SPECIFIC INDEX OF ARRAY Ap[] AND GO UNTIL THE NEXT INDEX OF THE ARRAY
		//WE PUT 1 TO THE INDEX OF MARK ARRAY WHERE WE FIND AN EDGE SO THAT WE CAN COUNT THE EDGES NEXT
		//USDE THE id VARIABLE BECAUSE id VARIABLE CONTAINS blockidx.x, threadidx.x, threadidx.y SO THAT EACH THREAD IS DOING INDIVIDUAL REGION OF EACH COLUMN
        for (int p = Ap[j]+id  ; p< Ap[j+1] ; p=p+NTHREADS)  //TODO
        {
            int i = Ai [p] ;
            /* edge (i,j) exists in the graph, mark node i */
            Mark [i] = 1 ;
        }
        __syncthreads( );
        /* compute sum(C(:,j)) where C=(A*A(:,j))*.(A(:,j)) */

        //THESE TWO FOR LOOPS ARE THE MEAT OF THE PROJECT
		//THE OUTER FOR LOOP GOING X DIRECTION AND THE INNER FOR LOOP GOING Y DIRECTION OF THE MATRIX
		//FOR EACH NODE FOUND IN THE COLUMN(OUTER LOOP), WE SEE HOW MANY MANY FRIEND THIS NODE HAS
		//FOR EACH FRIENDS, WE SEE IF THAT FRIEND IS ALSO THE CURRENT NODE'S FRIEND(INNER LOOP)
		//IF A FRIEND OF FRIEND IS ALSO MY FRIENDS, THAT MAKES A TRIANGLE, WE INCREMENT nt VARIABLE IF WE DETECT SUCH PHENOMENON
        for (int p = Ap[j]+threadIdx.x  ; p< Ap[j+1] ; p=p+NX)   //TODO
        {
            int k = Ai [p] ;
            /* edge (k,j) exists in the graph */
            for (int pa = Ap[k]+threadIdx.y ; pa < Ap[k+1] ; pa=pa+NY)  //TODO
            {
                int i = Ai [pa] ;
                /* edge (i,k) exists, count a triangle if (i,j) also exists */
                nt += Mark [i] ;
            }
        }

		__syncthreads ( ) ;
		
        /* clear Mark for the next iteration */
		//WE CLEAR THE Mark[] ARRAY SAME WAY WE FILLED IT UP 
		//THIS FOR LOOP IS EXACTLY THE SAME AS ABOVE
        for (int p = Ap[j]+id  ; p< Ap[j+1] ; p=p+NTHREADS)  //TODO
        {
            int i = Ai [p] ;
            /* edge (i,j) exists in the graph, mark node i */
            Mark [i] = 0 ;
        }

		__syncthreads ( ) ;
		
        /* now all of Mark[0..n-1] is all zero again */
        
        // only a few of the entries in Mark have been used in this jth
        // iteration.
    }
	
    /* each thread copies its result, nt, into a shared array, Ntri */
    // Ntri is a shared array of size Ntri[blockDim.y][blockDim.x] ; but size
    // must be constant so NY and NX are used.  Every thread saves its triangle
    // count in this __shared__ array so the results can be summed up for this
    // threadblock.  This part is done for you:
    __shared__ int Ntri [NY][NX] ;
    Ntri [threadIdx.y][threadIdx.x] = nt ;
    __syncthreads ( ) ;

    /* sum up all of Ntri and then one thread writes result to */
    /* Ntri_result [blockIdx.x] */
    // Now sum up all the triangles found by this threadblock,
    // Ntri_result [blockIdx.x] = sum (Ntri).  In your first attempt,
    // I recommend using thread (0,0) to do this work all by itself.
    // But don't stop there, do this reduction in parallel.
    // Figure this out yourself.
    //TODO
	//THIS REUCTION IS DONE IN PARALLEL ONLY BY THREAD 0,0
	//THIS TAKES ALL THE COUNTED TRIANGLE BY EACH NODE AND PUTS THEM IN Ntr_result[] WHICH KEEPS COUNT FOR EACH BLOCK
	if(id==0){
		Ntri_result[blockIdx.x] = 0;
		for(int y =0 ;y < NY; y++){
			for(int x =0; x < NX ; x++ ){
				Ntri_result [blockIdx.x]+= Ntri [y][x];
			}
		}
		
		__syncthreads ( ) ;
	//printf("-----The Ntri_result[] for block %d is %i \n",blockIdx.x , Ntri_result[blockIdx.x]);
	}

	
}


/* call a cuda method and check its error code */
// This is written for you already.
#define OK(method)                                          \
{                                                           \
    err = method ;                                          \
    if (err != hipSuccess)                                 \
    {                                                       \
        printf ("ERROR: line %d\n%s\n", __LINE__,           \
            hipGetErrorString (err)) ;                     \
        exit (1) ;                                          \
    }                                                       \
}

/* -------------------------------------------------------------------------- */
/* tri_gpu: driver function that runs on the host CPU */
/* -------------------------------------------------------------------------- */

int64_t tri_gpu         /* # of triangles                       */
(
    const int *Ap,      /* node pointers, size n+1              */
    const int *Ai,      /* adjacency lists, size ne = Ap [n]    */
    const int n         /* number of nodes in the graph         */
)
{
    hipError_t err = hipSuccess ;

    /* allocate the graph on the GPU */
    // This is written for you already.
    int ne = Ap [n] ;
    int *d_Ap, *d_Ai ;
    OK (hipMalloc (&d_Ap, (n+1) * sizeof (int))) ;
    OK (hipMalloc (&d_Ai, (ne ) * sizeof (int))) ;

    /* copy the graph to the GPU */
    //COPYING THE GRAPH FORM THE CPU TO THE GPU
    OK (hipMemcpy (d_Ap,   Ap,   (n+1) * sizeof(int),   hipMemcpyHostToDevice));  //TODO
    OK (hipMemcpy (d_Ai,   Ai,   (ne ) * sizeof(int),   hipMemcpyHostToDevice));	//TODO

    /* allocate workspace on the GPU */
    /* Marks array of size NBLOCKS * n * sizeof (bool), so each */
    /* threadblock has its own bool Mark array of size n.       */
    bool *d_Marks ;
    // CREATING d_Marks ARRAY OF SIZE NBLOKS*n*sizeof(bool) IN THE GPU
    OK (hipMalloc (&d_Marks, (NBLOCKS * n * sizeof (bool)))) ;	//TODO

    /* allocate the result on the GPU */
    int64_t *d_ntri ;
    // USING CUDAMALLOC TO ALLOCATE THE D_NTRI RESULT ON THE GPU, OF SIZE NBLOCKS
    OK (hipMalloc (&d_ntri, (NBLOCKS*sizeof(int64_t))));	//TODO

    // start the timer (optional, if you want to time just the kernel):
    // hipEvent_t start, stop ;
    // OK (hipEventCreate (&start)) ;
    // OK (hipEventCreate (&stop)) ;
    // OK (hipEventRecord (start)) ;

    /* launch the kernel */
    // this is written for you
    tri_kernel <<<NBLOCKS, dim3(NX,NY)>>> (d_Ap, d_Ai, n, d_Marks, d_ntri) ;
    OK (hipGetLastError ( )) ;

    // stop the timer (optional, if you want to time just the kernel)
    // OK (hipEventRecord (stop)) ;
    // OK (hipEventSynchronize (stop)) ;
    // float milliseconds = 0;
    // OK (hipEventElapsedTime (&milliseconds, start, stop)) ;
    // printf ("GPU kernel time: %g sec\n", milliseconds / 1000) ;

    /* get the result from the GPU: one value for each threadblock */
    int64_t ntri = 0, ntris [NBLOCKS] ;
    // GETTING THE D_NTRI ARRAY OF SIZE NBLOCKS FROM THE GPU
    OK (hipMemcpy (ntris, d_ntri, (NBLOCKS*sizeof(int64_t)), hipMemcpyDeviceToHost ));	//TODO

    /* free space on the GPU */
    // use hipFree to free all the things you hipMalloc'd.
    // if you fail to do this some problems will run out of memory
	//FREEING ALL THE MEMORY I HAVE ALLOCATED
	//NOTE: I DIDNT FREE d_ntri AND d_Marks ARRAY HERE COZ THERE WERE NO "TODO" TO DO THAT
    OK (hipFree (d_Ap)) ;
    OK (hipFree (d_Ai ));	//TODO

    /* sum up the results for all threadblocks */
    // the host has the result of each threadblock in ntris[NBLOCKS].
    // sum them up here into ntri.
    //TODO
	//FINAL REDUCTION OF ALL TRIANGLE COUNT FOR ALL BLOCKS
	ntri = 0;	
	for(int x = 0 ; x < NBLOCKS ; x++){
		ntri+= ntris[x];
	}

    /* return the result */
    return (ntri) ;
}
